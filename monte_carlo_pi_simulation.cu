#include "hip/hip_runtime.h"
/*
Simple Monte Carlo Pi Simulation using CUDA Primatives
*/

#include <hiprand.h>
#include <iostream>
#include <iomanip>

__device__ int total_device_points{};

__global__ void measure_points(const float* random_x,
	const float* random_y)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const float x = random_x[i] - 0.5F;
	const float y = random_y[i] - 0.5F;
	const int n = sqrtf(pow(x, 2) + pow(y, 2)) > 0.5F ? 0 : 1;
	atomicAdd(&total_device_points, n);

}
int main() {
	constexpr int width = 512;
	constexpr int height = 512;
	constexpr int count = width * height;
	constexpr int size = count * sizeof(float);

	hiprandGenerator_t random_generator;

	hiprandCreateGenerator(&random_generator, HIPRAND_RNG_PSEUDO_MTGP32);
	hiprandSetPseudoRandomGeneratorSeed(random_generator, time(nullptr));

	float *random_x, *random_y;
	hipMalloc(&random_x, size);
	hipMalloc(&random_y, size);

	hiprandGenerateUniform(random_generator, random_x, count);
	hiprandGenerateUniform(random_generator, random_y, count);

    measure_points << <width, height >> > (random_x, random_y);
    
	int total_host_points;
	hipMemcpyFromSymbol(&total_host_points, HIP_SYMBOL(total_device_points), sizeof(int));

	const float estimated_pi = ((4.0F * static_cast<float>(total_host_points)) / static_cast<float>(count));

	std::cout << std::setprecision(std::numeric_limits<float>::digits10 + 1)
		<< "Using the Monte Carlo Method Pi is estimated to be: "
		<< estimated_pi
		<< '\n';

	hipFree(random_x);
	hipFree(random_y);
}